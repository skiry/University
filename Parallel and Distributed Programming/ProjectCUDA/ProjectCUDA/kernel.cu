#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <string>
#include <chrono>
#include "utils.h"
#include "timer.h"

void processCudaGrayscale(std::string input_file, std::string output_file);

__global__ void rgbaToGreyscaleCudaKernel(const uchar4* const rgbaImage,
    unsigned char* const greyImage,
    const int numRows, const int numCols)
{
    /*
    To convert an image from color to grayscale one simple method is to
    set the intensity to the average of the RGB channels.  But we will
    use a more sophisticated method that takes into account how the eye 
    perceives color and weights the channels unequally.

    The eye responds most strongly to green followed by red and then blue.
    The NTSC (National Television System Committee) recommends the following
    formula for color to greyscale conversion:

    I = .299f * R + .587f * G + .114f * B*/

    //First create a mapping from the 2D block and grid locations
    //to an absolute 2D location in the image, then use that to
    //calculate a 1D offset
    const long pointIndex = threadIdx.x + blockDim.x*blockIdx.x;

    if (pointIndex < numRows*numCols) { // this is necessary only if too many threads are started
        uchar4 const imagePoint = rgbaImage[pointIndex];
        greyImage[pointIndex] = .299f*imagePoint.x + .587f*imagePoint.y + .114f*imagePoint.z;
    }
}

// Parallel implementation for running on GPU using multiple threads.
void rgbaToGreyscaleCuda(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage,
    unsigned char* const d_greyImage, const size_t numRows, const size_t numCols)
{
    const int blockThreadSize = 512;
    const int numberOfBlocks = 1 + ((numRows*numCols - 1) / blockThreadSize); // a/b rounded up
    const dim3 blockSize(blockThreadSize, 1, 1);
    const dim3 gridSize(numberOfBlocks, 1, 1);
    rgbaToGreyscaleCudaKernel << <gridSize, blockSize >> > (d_rgbaImage, d_greyImage, numRows, numCols);
}

int main()
{
    std::string input_file = "cutzu.jpg";
    std::string output_cuda_file_grayscale = "cutzu-grayscale.bmp";

    processCudaGrayscale(input_file, output_cuda_file_grayscale);

    cleanupCuda();

    return 0;
}

void processCudaGrayscale(std::string input_file, std::string output_file) {
    // pointers to images in CPU's memory (h_) and GPU's memory (d_)
    uchar4        *h_rgbaImage, *d_rgbaImage;
    unsigned char *h_greyImage, *d_greyImage;

    //load the image and give us our input and output pointers
    preProcess(&h_rgbaImage, &h_greyImage, &d_rgbaImage, &d_greyImage, input_file);

    GpuTimer timer;
    timer.Start();
    // here is where the conversion actually happens
    rgbaToGreyscaleCuda(h_rgbaImage, d_rgbaImage, d_greyImage, numRows(), numCols());
    timer.Stop();
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

    int err = printf("Implemented CUDA code ran in: %f msecs.\n", timer.Elapsed());

    if (err < 0) {
        //Couldn't print!
        std::cerr << "Couldn't print timing information! STDOUT Closed!" << std::endl;
        exit(1);
    }

    size_t numPixels = numRows()*numCols();
    checkCudaErrors(hipMemcpy(h_greyImage, d_greyImage, sizeof(unsigned char) * numPixels, hipMemcpyDeviceToHost));

    //check results and output the grey image
    postProcess(output_file, h_greyImage);
}

