#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define RUN_KARATSUBA 1

static void HandleError(hipError_t err,
    const char *file,
    int line) {
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err),
            file, line);
        exit(EXIT_FAILURE);
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

hipError_t classicalCuda(int *c, const int *a, const int *b, unsigned int size);
hipError_t karatsubaCuda(int *c, const int *a, const int *b, unsigned int size);
hipError_t karatsubaCudaCorrect(int *c, const int *a, const int *b, unsigned int size);

__device__ void multiplyKernelKaratsubaRec(int *z, const int *x, const int *y, const int size)
{
    const int *a, *b, *c, *d;
    int *ab, *ac;
    int *bd, *cd;
    int *adbc;


    if (size <= 1)
    {
        z[0] = x[0] * y[0];
    }
    else
    {
        int half = (int)size / 2;

        ab = (int*)malloc(half * sizeof(int));
        ac = (int*)malloc(half * sizeof(int));
        cd = (int*)malloc(half * sizeof(int));
        bd = (int*)malloc(half * sizeof(int));
        adbc = (int*)malloc(half * sizeof(int));

        a = x;
        b = x + half;

        c = y;
        d = y + half;

        multiplyKernelKaratsubaRec(ac, a, c, half);
        multiplyKernelKaratsubaRec(bd, b, d, size - half);

        int i = 0;
        for (i = 0; i < half; i++)
        {
            ab[i] = a[i] + b[i];
            cd[i] = c[i] + d[i];
        }

        multiplyKernelKaratsubaRec(adbc, ab, cd, half);

        for (i = 0; i < half; i++)
        {
            z[i] = adbc[i] - ac[i] - bd[i];
        }
    }
}

__global__ void multiplyKernelKaratsuba(int *z, const int *x, const int *y, const int size)
{
    multiplyKernelKaratsubaRec(z, x, y, size);
}

__global__ void multiplyKernel(int *c, const int *a, const int *b, const int size)
{
    int i = threadIdx.x;
    for (int j = 0; j < size; ++j) {
        c[i + j] += a[i] * b[j];
    }
}


int* generatePolynome(const int size)
{
    int *pol = (int*)malloc(size * sizeof(int));
    for (int i = 0; i < size; i++)
        pol[i] = rand() % 2020;
    return pol;
}

int main()
{
    const int arraySize = 64000;
    const int* a = generatePolynome(arraySize);
    const int* b = generatePolynome(arraySize);
    int c[2 * arraySize] = { 0 };

    /*
    float time;
    hipEvent_t start, stop;

    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));
    HANDLE_ERROR(hipEventRecord(start, 0));
    */
    time_t timeStart;
    time_t timeEnd;

    time(&timeStart);

    hipError_t cudaStatus;

    if (RUN_KARATSUBA) {
        // Multiply vectors in parallel with the karatsuba algorithm
        cudaStatus = karatsubaCudaCorrect(c, a, b, arraySize);
    }
    else {
        // Multiply vectors in parallel with the n squared algorithm
        cudaStatus = classicalCuda(c, a, b, arraySize);
    }

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "multiplyWithCuda failed!");
        return 1;
    }

    time(&timeEnd);
    /*
    HANDLE_ERROR(hipEventRecord(stop, 0));
    HANDLE_ERROR(hipEventSynchronize(stop));
    HANDLE_ERROR(hipEventElapsedTime(&time, start, stop));
    */
    printf("Time to multiply:  %3.1lld ms \n", timeEnd - timeStart);

    /*for (int i = 0; i < arraySize; i++)
        printf("%d ", a[i]);

    printf("\n\n");

    for (int i = 0; i < arraySize; i++)
        printf("%d ", b[i]);

    printf("\n\n");

    for (int i = 0; i < 2 * arraySize - 1; i++)
        printf("%d ", c[i]);

    printf("\n");*/

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to multiply vectors in parallel.
hipError_t classicalCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, 2 * size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    multiplyKernel<<<1, size>>>(dev_c, dev_a, dev_b, size);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "multiplydKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching multiplyKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, 2 * size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}

hipError_t karatsubaCudaCorrect(int *c, const int *a, const int *b, unsigned int size) {
    int *dev_a = nullptr;
    int *dev_b = nullptr;
    int *dev_c = nullptr;
    hipError_t cudaStatus;

    hipSetDevice(0);

    hipMalloc(&dev_c, 2 * size * sizeof(int));
    hipMalloc(&dev_a, size * sizeof(int));
    hipMalloc(&dev_b, size * sizeof(int));

    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    int thread_num = 2 * size;
    multiplyKernelKaratsuba << <1, thread_num >> > (dev_c, dev_a, dev_b, size);

    cudaStatus = hipDeviceSynchronize();

    hipMemcpy(c, dev_c, 2 * size * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return hipSuccess;
}

// Helper function for using CUDA to multiply vectors in parallel.
hipError_t karatsubaCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, 2 * size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    multiplyKernelKaratsuba << <1, 2 * size >> > (dev_c, dev_a, dev_b, size);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "multiplydKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching multiplyKernelKaratsuba!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, 2 * size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}
